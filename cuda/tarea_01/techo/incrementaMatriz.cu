#include "hip/hip_runtime.h"
//
//  sumaMatrices.cu
//  
//
//  Created by Amilcar Meneses Viveros on 15/02/18.
//
//

#include <stdio.h>

#define M 8192
#define N 8192 

int nbx, nby, ntx, nty;

double a[M][N], b[M][N], c[M][N];

__global__ void kernelIncrementaMatriz(double *b, double *c, int m, int n) {
       int i = threadIdx.x+blockIdx.x*blockDim.x;
       int j = threadIdx.y+blockIdx.y*blockDim.y;

       while (i<m) {
         j = threadIdx.y+blockIdx.y*blockDim.y;
         while (j<n) {
             b[i*n+j]+=c[i*n+j];
             j += blockDim.x*gridDim.x; 
         }
         i+=blockDim.y*gridDim.y;
      }
}

void sumaMatricesEnDevice(double a[][N], double b[][N], double c[][N], int m, int n) {
    int size=m*n*sizeof(double);
    double *bD, *cD;
    dim3 nb(nbx, nby);   //16
    dim3 nt(ntx, nty); //1024 = 16000 hilos
   
    hipSetDevice(0);  
    // 1. Reservar memoria
    hipMalloc(&bD, size);  hipMalloc(&cD, size);

    // 2. Subir datos del Host a Device
    hipMemcpy(bD, b, size, hipMemcpyDefault);
    hipMemcpy(cD, c, size, hipMemcpyDefault);

    // 3. Ejecutar kernel
    kernelIncrementaMatriz<<<nb, nt>>>(bD, cD, m, n);
    
    // 4. Bajar datos del Device al Host
    hipMemcpy(a, bD, size, hipMemcpyDefault);
 
   // 5.Libera memoria
    hipFree(bD); hipFree(cD);
}

int main() {
    int i, j;

    // Implementacion de techo.
    ntx = 32;
    nty = 32;
    nbx = cell(N/ntx);
    nby = cell(N/nty);

    for (i=0; i<M; i++) {
        for (j=0; j<N; j++) {
            b[i][j] = c[i][j] = i+j;
        }
    }
    sumaMatricesEnDevice(a, b, c, M, N);

    for (i=M-1; i<M; i++) {
        for (j=0; j<N; j++) {
            printf("%3.2lf ", a[i][j]);
        }
        printf("\n");
    }
}


//
//  traspuestaMatriz.cu
//  
//
//  Created by Amilcar Meneses Viveros on 15/02/18.
//
//


#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 10
#define N 10

double a[M][N], b[M][N], c[M][N];

__global__ void kernelIncrementaMatriz(double *b, double *c, int m, int n) {
       int i = threadIdx.x+blockIdx.x*blockDim.x;
       int j = threadIdx.y+blockIdx.y*blockDim.y;

       while (i<m) {
         j = threadIdx.y+blockIdx.y*blockDim.y;
         while (j<n) {
             b[i*n+j]+=c[i*n+j];
             j += blockDim.x*gridDim.x; 
         }
         i+=blockDim.y*gridDim.y;
      }
}

void traspuestaMatrizOnDevice(double a[][N], double b[][N], double c[][N], int m, int n) {
    int size=m*n*sizeof(double);
    double *bD, *cD;
    dim3 nb(2,2);
    dim3 nt(5,5);
   
    hipSetDevice(0);  
    // 1. Reservar memoria
    hipMalloc(&bD, size);  hipMalloc(&cD, size);

    // 2. Subir datos del Host a Device
    hipMemcpy(bD, b, size, hipMemcpyDefault);
    hipMemcpy(cD, c, size, hipMemcpyDefault);

    // 3. Ejecutar kernel
    kernelIncrementaMatriz<<<nb, nt>>>(bD, cD, m, n);
    
    // 4. Bajar datos del Device al Host
    hipMemcpy(a, bD, size, hipMemcpyDefault);
 
   // 5.Libera memoria
    hipFree(bD); hipFree(cD);
}

int main() {
    int i, j;
    for (i=0; i<M; i++) {
        for (j=0; j<N; j++) {
            b[i][j] = c[i][j] = i+j;
        }
    }
    traspuestaMatrizOnDevice(a, b, c, M, N);

    for (i=0; i<M; i++) {
        for (j=0; j<N; j++) {
            printf("%3.2lf ", a[i][j]);
        }
        printf("\n");
    }
}


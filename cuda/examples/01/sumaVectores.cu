


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10000 

float a[N], b[N], c[N]; 

__global__ void kernelSumaVectores(float *a, float *b, float *c, int n)  
{
    int i = threadIdx.x+blockIdx.x*blockDim.x;

    while (i<n) {
        c[i]=a[i]+b[i];
        i+= blockDim.x*gridDim.x; 
   }
}

void sumaVectoresEnDevice(float *a, float *b, float *c, int n) {
    float *aD, *bD, *cD; 
    int size=n*sizeof(float); 

    hipSetDevice(0); 

    // 1. Asignar memoria 
    hipMalloc(&aD, size); 
    hipMalloc(&bD, size); 
    hipMalloc(&cD, size); 
    
   // 2. Copiar datos del Host al Device 
   hipMemcpy(aD, a, size, hipMemcpyHostToDevice);    
   hipMemcpy(bD, b, size, hipMemcpyHostToDevice);    

   // 3. Ejecutar kernel 
   kernelSumaVectores<<<10, 1024>>>(aD, bD, cD, n);

   // 4. Copiar datos del device al Host
   hipMemcpy(c, cD, size, hipMemcpyDeviceToHost);    

   // 5. Liberar Memoria
   hipFree(aD); hipFree(bD); hipFree(cD); 
}

int main() {
   int i; 

  for (i=0; i<N; i++) { 
      a[i] = b[i] = i; 
  }

  sumaVectoresEnDevice(a, b, c, N); 

  for (i=0; i<N; i++) { 
      printf("c[%d] = %f\n", i, c[i]); 
  }

}




#include <hip/hip_runtime.h>
#include <stdio.h>
#include  <math.h>

// sin(x*x)*cos(x)*x -  x*x  

#define XMIN  0.0  
#define XMAX  10.0 
#define  N 10000 

double vh[N], vd[N]; 

__device__  __host__ double funcion(double x) {
   return sin(x*x)*cos(x)*x -  x*x; 
}

__global__ void kernelEvaluaFuncion(double *v, double xmin, double xmax, int n){
  int i=threadIdx.x + blockIdx.x*blockDim.x;  
  double dx= (xmax-xmin)/(double)(n-1);
  double x = xmin + i*dx;  
  v[i] = funcion(x); 
}


void llenaPolinomioEnDevice(double *v, int n) {
    double *vD; 
    int size = n*sizeof(double); 

    hipMalloc(&vD, size); 
    kernelEvaluaFuncion<<<10, 1000>>>(vD, XMIN, XMAX, n); 
    hipMemcpy(v, vD, size, hipMemcpyDeviceToHost); 
    hipFree(vD); 
}

void llenaPolinomioEnHost(double *v, int n) {
   int i; 
   double x, dx;
   dx = (XMAX-XMIN)/(double)(N-1);

   x = XMIN; 
   for (i=0; i<N; i++) {
      v[i] = funcion(x); 
      x+=dx;
   }
}

int main() {
   int i; 
   double x, dx; 
   dx = (XMAX-XMIN)/(double)(N-1); 

   llenaPolinomioEnHost(vh, N); 
   llenaPolinomioEnDevice(vd, N); 

   x = XMIN; 
   for (i=0; i<N; i++) {
      printf("%4d %4.2lf    %6.12lf    %6.12lf\n", i, x, vh[i], vd[i]); 
      x+=dx; 
   }
}


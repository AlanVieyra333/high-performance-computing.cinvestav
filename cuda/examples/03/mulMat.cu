//
//  mulMat.cu
//  
//
//  Created by Amilcar Meneses Viveros on 15/02/18.
//
//


#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 1000
#define N 1000 

// double a[M][N], b[M][N], c[M][N];
double *a, *b, *c; 
float elapsedtime_total;
float elapsedtime_kernel; 

__global__ void kernelmultiplicaMatrices(double *a, double *b, double *c, int m, int n) {
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k; 
    double tmp;    
 
    tmp = 0; 
    for (k=0; k<n; k++) {
        tmp += b[i*m+k]*c[k*m+j]; 
    }
   a[i*m+j] = tmp;     
}

void multiplicaMatricesEnDevice(double *a, double *b, double *c, int m, int n) {
    int size=m*n*sizeof(double);
    double *aD, *bD, *cD;
    dim3 nb(50,50);
    dim3 nt(20,20);
    hipEvent_t start, stop; 
    hipEvent_t startk, stopk;

    hipSetDevice(0);  
    hipEventCreate(&start); 
    hipEventCreate(&stop); 
    hipEventCreate(&startk);
    hipEventCreate(&stopk);

    hipEventRecord(start, 0); 

    // 1. Reservar memoria
    hipMalloc(&aD, size); hipMalloc(&bD, size);  hipMalloc(&cD, size);

    // 2. Subir datos del Host a Device
    hipMemcpy(bD, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cD, c, size, hipMemcpyHostToDevice);

    // 3. Ejecutar kernel
    hipEventRecord(startk, 0);
       kernelmultiplicaMatrices<<<nb, nt>>>(aD, bD, cD, m, n);
    hipEventRecord(stopk, 0); 
    hipEventSynchronize(stopk);

    // 4. Bajar datos del Device al Host
    hipMemcpy(a, aD, size, hipMemcpyDeviceToHost);
 
   // 5.Libera memoria
    hipFree(aD); hipFree(bD); hipFree(cD);

   hipEventRecord(stop, 0); 
   hipEventSynchronize(stop); 

   hipEventElapsedTime(&elapsedtime_total, start, stop); 
   hipEventElapsedTime(&elapsedtime_kernel, startk, stopk);

   hipEventDestroy(start); hipEventDestroy(stop); 
   hipEventDestroy(startk); hipEventDestroy(stopk);

}

int main() {
    int i, j;

    a = (double*)malloc(M*N*sizeof(double));
    b = (double*)malloc(M*N*sizeof(double)); 
    c = (double*)calloc(M*N, sizeof(double)); 

    for (i=0; i<M; i++) {
        for (j=0; j<N; j++) {
            b[i*N+j] = i+j;
        }
        c[i*N+i] = 2.0;

    }

    multiplicaMatricesEnDevice(a, b, c, M, N);
    for (i=M-1; i<M; i++) {
        for (j=0; j<N; j++) {
            printf("%3.2lf ", a[i*N+j]);
        }
        printf("\n");
    } 
    free(a); free(b); free(c); 

    printf("Tiempo total cuda %4.6fms\n", elapsedtime_total);
    printf("Tiempo kernel cuda %4.6fms\n", elapsedtime_kernel);  
    return 0; 
}


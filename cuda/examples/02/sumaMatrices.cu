//
//  sumaMatrices.cu
//  
//
//  Created by Amilcar Meneses Viveros on 15/02/18.
//
//


#include <hip/hip_runtime.h>
#include <stdio.h>

#define M 8192
#define N 8192 

double a[M][N], b[M][N], c[M][N];

__global__ void kernelSumaMatrices(double *a, double *b, double *c, int m, int n) {
       int i = threadIdx.x+blockIdx.x*blockDim.x;
       int j = threadIdx.y+blockIdx.y*blockDim.y;

       while (i<m) {
         j = threadIdx.y+blockIdx.y*blockDim.y;
         while (j<n) {
             a[i*n+j] = b[i*n+j]+c[i*n+j];
             j += blockDim.x*gridDim.x; 
         }
         i+=blockDim.y*gridDim.y;
      }
}

void sumaMatricesEnDevice(double a[][N], double b[][N], double c[][N], int m, int n) {
    int size=m*n*sizeof(double);
    double *aD, *bD, *cD;
    dim3 nb(4,4);
    dim3 nt(32,32);
   
    hipSetDevice(0);  
    // 1. Reservar memoria
    hipMalloc(&aD, size); hipMalloc(&bD, size);  hipMalloc(&cD, size);

    // 2. Subir datos del Host a Device
    hipMemcpy(bD, b, size, hipMemcpyDefault);
    hipMemcpy(cD, c, size, hipMemcpyDefault);

    // 3. Ejecutar kernel
    kernelSumaMatrices<<<nb, nt>>>(aD, bD, cD, m, n);
    
    // 4. Bajar datos del Device al Host
    hipMemcpy(a, aD, size, hipMemcpyDefault);
 
   // 5.Libera memoria
    hipFree(aD); hipFree(bD); hipFree(cD);
}

int main() {
    int i, j;
    for (i=0; i<M; i++) {
        for (j=0; j<N; j++) {
            b[i][j] = c[i][j] = i+j;
        }
    }
    sumaMatricesEnDevice(a, b, c, M, N);

    for (i=M-1; i<M; i++) {
        for (j=0; j<N; j++) {
            printf("%3.2lf ", a[i][j]);
        }
        printf("\n");
    }
}

